
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>


void cpu_conv1D(float *a, float *b, float *c, int size){
	int temp = 0;
        //If index less than 0 or more or equal than N, set 0 as the array element
	for (int i = 0; i < size; i++){
	    for (int j = -1; j < 2; j++){
                if ( ( i+j > 0 ) || ( i+j < size ) ){
                    temp += a[i+j]*b[i+j];
                }
            }
            c[i] = temp;
            temp = 0;  
	}
}

/*
void cpu_conv2D(float *
*/

void fillArray(float * m, int size, float v){
	for (int i = 0; i < size; i++){
		m[i] = v;
	}
}

void printArray(float *m, int size){
    for (int i = 0; i < size; i++){
        printf("%f ", m[i]);
    }
    printf("\n");
}

int main(int argc, char *argv[]){
	
	//set size as an integer number
	int size;
	//set a, b and c arrays
	float *a, *b, *c;
	if (argc == 2) {
		size = atoi(argv[1]);
        	printf("size: %d\n", size);		

		a = (float *) malloc(size*sizeof(float));
		b = (float *) malloc(size*sizeof(float));
		c = (float *) malloc(size*sizeof(float));
		
		
		/*
		Solo hacer multiplicaciones de elementos
		Que no se encuentren dentro de los límites 
		de los arreglos
		*/
		
		
		//llenar arreglo
		fillArray(a, size, 2);
		fillArray(b, size, 1);
		fillArray(c, size, 0);
	
        	printf("a: ");
        	printArray(a, size);
        	printf("b: ");
        	printArray(b, size);
        	printf("c: ");
        	printArray(c, size);
	
		cpu_conv1D(a,b,c,size);
        
        	printf("New c: ");
        	printArray(c, size);        
        
        	free(a);
        	free(b);
        	free(c);
	}
	
	
	return 0;
}

